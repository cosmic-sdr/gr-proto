
#include <hip/hip_runtime.h>
#ifndef __CUDA_KERNELHEADER__ 
#define __CUDA_KERNELHEADER__ 
/********************************************/
/* Added codes for OpenACC2CUDA translation */
/********************************************/
#ifdef __cplusplus
#define restrict __restrict__
#endif
#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38
#endif
#ifndef FLT_MIN
#define FLT_MIN 1.175494351e-38
#endif
#ifndef DBL_MAX
#define DBL_MAX 1.7976931348623158e+308
#endif
#ifndef DBL_MIN
#define DBL_MIN 2.2250738585072014e-308
#endif
#endif


extern "C" __global__ void accLog_kernel_kernel0(float * in1, float * out, float k_val, float n_val, int noutput_items)
{
int lwpriv__i;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
out[lwpriv__i]=((n_val*log10(in1[lwpriv__i]))+k_val);
}
}

#ifndef __CUDA_KERNELHEADER__ 
#define __CUDA_KERNELHEADER__ 
/********************************************/
/* Added codes for OpenACC2CUDA translation */
/********************************************/
#ifdef __cplusplus
#define restrict __restrict__
#endif
#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38
#endif
#ifndef FLT_MIN
#define FLT_MIN 1.175494351e-38
#endif
#ifndef DBL_MAX
#define DBL_MAX 1.7976931348623158e+308
#endif
#ifndef DBL_MIN
#define DBL_MIN 2.2250738585072014e-308
#endif
#endif


struct FComplexStruct
{
float real;
float imag;
};

typedef struct FComplexStruct FComplex;
extern "C" __global__ void accComplexToArg_kernel_kernel0(FComplex * in, float * out, int noutput_items)
{
int lwpriv__i;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
out[lwpriv__i]=atan2(in[lwpriv__i].imag, in[lwpriv__i].real);
}
}

#ifndef __CUDA_KERNELHEADER__ 
#define __CUDA_KERNELHEADER__ 
/********************************************/
/* Added codes for OpenACC2CUDA translation */
/********************************************/
#ifdef __cplusplus
#define restrict __restrict__
#endif
#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38
#endif
#ifndef FLT_MIN
#define FLT_MIN 1.175494351e-38
#endif
#ifndef DBL_MAX
#define DBL_MAX 1.7976931348623158e+308
#endif
#ifndef DBL_MIN
#define DBL_MIN 2.2250738585072014e-308
#endif
#endif


extern "C" __global__ void accComplexToMag_kernel_kernel0(FComplex * in, float * out, int noutput_items)
{
int lwpriv__i;
float aval;
float bval;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
aval=in[lwpriv__i].imag;
bval=in[lwpriv__i].real;
out[lwpriv__i]=sqrt(((aval*aval)+(bval*bval)));
}
}

#ifndef __CUDA_KERNELHEADER__ 
#define __CUDA_KERNELHEADER__ 
/********************************************/
/* Added codes for OpenACC2CUDA translation */
/********************************************/
#ifdef __cplusplus
#define restrict __restrict__
#endif
#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38
#endif
#ifndef FLT_MIN
#define FLT_MIN 1.175494351e-38
#endif
#ifndef DBL_MAX
#define DBL_MAX 1.7976931348623158e+308
#endif
#ifndef DBL_MIN
#define DBL_MIN 2.2250738585072014e-308
#endif
#endif


extern "C" __global__ void accComplexToMagPhase_kernel_kernel0(FComplex * in, float * out0, float * out1, int noutput_items)
{
int lwpriv__i;
float aval;
float bval;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
aval=in[lwpriv__i].imag;
bval=in[lwpriv__i].real;
out0[lwpriv__i]=sqrt(((aval*aval)+(bval*bval)));
out1[lwpriv__i]=atan2(aval, bval);
}
}

#ifndef __CUDA_KERNELHEADER__ 
#define __CUDA_KERNELHEADER__ 
/********************************************/
/* Added codes for OpenACC2CUDA translation */
/********************************************/
#ifdef __cplusplus
#define restrict __restrict__
#endif
#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38
#endif
#ifndef FLT_MIN
#define FLT_MIN 1.175494351e-38
#endif
#ifndef DBL_MAX
#define DBL_MAX 1.7976931348623158e+308
#endif
#ifndef DBL_MIN
#define DBL_MIN 2.2250738585072014e-308
#endif
#endif


extern "C" __global__ void accComplexToMagSquared_kernel_kernel0(FComplex * in, float * out, int noutput_items)
{
int lwpriv__i;
float aval;
float bval;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
aval=in[lwpriv__i].imag;
bval=in[lwpriv__i].real;
out[lwpriv__i]=((aval*aval)+(bval*bval));
}
}

#ifndef __CUDA_KERNELHEADER__ 
#define __CUDA_KERNELHEADER__ 
/********************************************/
/* Added codes for OpenACC2CUDA translation */
/********************************************/
#ifdef __cplusplus
#define restrict __restrict__
#endif
#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38
#endif
#ifndef FLT_MIN
#define FLT_MIN 1.175494351e-38
#endif
#ifndef DBL_MAX
#define DBL_MAX 1.7976931348623158e+308
#endif
#ifndef DBL_MIN
#define DBL_MIN 2.2250738585072014e-308
#endif
#endif


extern "C" __global__ void accMagPhaseToComplex_kernel_kernel0(float * a, float * b, FComplex * c, int noutput_items)
{
int lwpriv__i;
float mag;
float phase;
float real;
float imag;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
mag=a[lwpriv__i];
phase=b[lwpriv__i];
real=mag*cos(phase);
imag=mag*sin(phase);
c[lwpriv__i].real=real;
c[lwpriv__i].imag=imag;
}
}

