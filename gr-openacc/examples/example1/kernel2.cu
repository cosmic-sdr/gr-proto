
#include <hip/hip_runtime.h>
#ifndef __CUDA_KERNELHEADER__ 
#define __CUDA_KERNELHEADER__ 
/********************************************/
/* Added codes for OpenACC2CUDA translation */
/********************************************/
#ifdef __cplusplus
#define restrict __restrict__
#endif
#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38
#endif
#ifndef FLT_MIN
#define FLT_MIN 1.175494351e-38
#endif
#ifndef DBL_MAX
#define DBL_MAX 1.7976931348623158e+308
#endif
#ifndef DBL_MIN
#define DBL_MIN 2.2250738585072014e-308
#endif
#endif


extern "C" __global__ void accLog_kernel_kernel0(float * in, float * out, float k_val, float n_val, int noutput_items)
{
int lwpriv__i;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
out[lwpriv__i]=((n_val*log10(in[lwpriv__i]))+k_val);
}
}

struct FComplexStruct
{
float real;
float imag;
};

typedef struct FComplexStruct FComplex;
__device__ static float dev__fast_atan2f_GP0_TU0_CT0(float y, float x, float fast_atan_table[])
{
float x_abs;
float y_abs;
float z;
float alpha;
float angle;
float base_angle;
int index;
/* normalize to +- 45 degree range */
float _ret_val_0;
y_abs=fabsf(y);
x_abs=fabsf(x);
/* don't divide by zero! */
if ( ! ((y_abs>0.0F)||(x_abs>0.0F)))
{
_ret_val_0=0.0;
return _ret_val_0;
}
if (y_abs<x_abs)
{
z=(y_abs/x_abs);
}
else
{
z=(x_abs/y_abs);
}
/* when ratio approaches the table resolution, the angle is */
/* best approximated with the argument itself... */
/* (smallest non-zero value in table) */
if (z<0.003921569)
{
base_angle=z;
}
else
{
/* find index and interpolation value */
alpha=(z*((float)255));
index=(((int)alpha)&0xff);
alpha-=((float)index);
/* determine base angle based on quadrant and */
/* add or subtract table value from base angle based on quadrant */
base_angle=fast_atan_table[index];
base_angle+=((fast_atan_table[(index+1)]-fast_atan_table[index])*alpha);
}
if (x_abs>y_abs)
{
/* -45 -> 45 or 135 -> 225 */
if (x>=0.0)
{
/* -45 -> 45 */
/* 0 -> 45, angle OK */
if (y>=0.0)
{
angle=base_angle;
}
else
{
angle=( - base_angle);
}
/* -45 -> 0, angle = -angle */
}
else
{
/* 135 -> 180 or 180 -> -135 */
angle=3.141592653589793;
/* 135 -> 180, angle = 180 - angle */
if (y>=0.0)
{
angle-=base_angle;
}
else
{
angle=(base_angle-angle);
}
/* 180 -> -135, angle = angle - 180 */
}
}
else
{
/* 45 -> 135 or -135 -> -45 */
if (y>=0.0)
{
/* 45 -> 135 */
angle=1.5707963267948966;
/* 45 -> 90, angle = 90 - angle */
if (x>=0.0)
{
angle-=base_angle;
}
else
{
angle+=base_angle;
}
/* 90 -> 135, angle = 90 + angle */
}
else
{
/* -135 -> -45 */
angle=( - 1.5707963267948966);
/* -90 -> -45, angle = -90 + angle */
if (x>=0.0)
{
angle+=base_angle;
}
else
{
angle-=base_angle;
}
/* -135 -> -90, angle = -90 - angle */
}
}
return angle;
}

extern "C" __global__ void accComplexToArg_kernel_kernel0(FComplex * in, float * out, float fast_atan_table[], int noutput_items)
{
int lwpriv__i;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
out[lwpriv__i]=dev__fast_atan2f_GP0_TU0_CT0(in[lwpriv__i].imag, in[lwpriv__i].real, fast_atan_table);
}
}

extern "C" __global__ void accComplexToMag_kernel_kernel0(FComplex * in, float * out, int noutput_items)
{
int lwpriv__i;
float aval;
float bval;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
aval=in[lwpriv__i].imag;
bval=in[lwpriv__i].real;
out[lwpriv__i]=sqrt(((aval*aval)+(bval*bval)));
}
}

extern "C" __global__ void accComplexToMagPhase_kernel_kernel0(FComplex * in, float * out0, float * out1, float fast_atan_table[], int noutput_items)
{
int lwpriv__i;
float aval;
float bval;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
aval=in[lwpriv__i].imag;
bval=in[lwpriv__i].real;
out0[lwpriv__i]=sqrt(((aval*aval)+(bval*bval)));
out1[lwpriv__i]=dev__fast_atan2f_GP0_TU0_CT0(aval, bval, fast_atan_table);
}
}

extern "C" __global__ void accComplexToMagSquared_kernel_kernel0(FComplex * in, float * out, int noutput_items)
{
int lwpriv__i;
float aval;
float bval;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
aval=in[lwpriv__i].imag;
bval=in[lwpriv__i].real;
out[lwpriv__i]=((aval*aval)+(bval*bval));
}
}

extern "C" __global__ void accMagPhaseToComplex_kernel_kernel0(float * a, float * b, FComplex * c, int noutput_items)
{
int lwpriv__i;
float mag;
float phase;
float real;
float imag;
lwpriv__i=(threadIdx.x+(blockIdx.x*64));
if (lwpriv__i<noutput_items)
{
mag=a[lwpriv__i];
phase=b[lwpriv__i];
real=mag*cos(phase);
imag=mag*sin(phase);
c[lwpriv__i].real=real;
c[lwpriv__i].imag=imag;
}
}

