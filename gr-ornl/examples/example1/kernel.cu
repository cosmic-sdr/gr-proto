
#include <hip/hip_runtime.h>
extern "C" __global__ void vecadd(int* out, int* in0, int* in1, size_t blockOff_x) {
  size_t id = (blockOff_x + blockIdx.x) * blockDim.x + threadIdx.x;
  out[id] = in0[id] + in1[id];
}

extern "C" __global__ void saxpy(int* out, int in0, int* in1, int* in2, size_t blockOff_x) {
  size_t id = (blockOff_x + blockIdx.x) * blockDim.x + threadIdx.x;
  out[id] = in0 * in1[id] + in2[id];
}

